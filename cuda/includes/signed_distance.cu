#include "hip/hip_runtime.h"
#pragma once

#include "../includes/libraries/glm/glm.hpp"
#include "../includes/types.cu"
#include "../includes/utils.cu"

using namespace glm;

__device__ float wrap(const float x, const float lower, const float higher) {
    return lower + glm::mod(x - lower, higher - lower);
}

__device__ vec3 wrap(const vec3 p, const vec3 lower, const vec3 higher) {
    return {
        wrap(p.x, lower.x, higher.x), wrap(p.y, lower.y, higher.y),
        wrap(p.z, lower.z, higher.z)
    };
}

__device__ float smooth_min(const float a, const float b, const float k) {
    float h = max(k - abs(a - b), 0.0f) / k;
    return min(a, b) - h * h * h * k * (1.0f / 6.0f);
}

// fractals

#define POWER 7.0f

__device__ float sd_mandelbulb(const vec3 p, const float time) {
    vec3 z = p;
    float dr = 1.0f;
    float r;

    float power = POWER * (1.0f + time * 0.001f);

    for (int i = 0; i < 25; i++) {
        r = length(z);
        if (r > 2.0f) {
            break;
        }

        float theta = acos(z.z / r) * power;
        float phi = atan2(z.y, z.x) * power;
        float zr = pow(r, power);
        dr = pow(r, power - 1.0f) * power * dr + 1.0f;

        float s_theta = sin(theta);
        z = zr * vec3(s_theta * cos(phi), sin(phi) * s_theta, cos(theta));
        z += p;
    }

    return 0.5f * log(r) * r / dr;
}

__device__ float sd_unit_mandelbulb(const vec3 p) {
    return sd_mandelbulb(p / 0.4f, 0.0f) * 0.4f;
}

// primitives

__device__ float sd_ray(const vec3 p, const vec3 bl, const vec3 bd) {
    return distance(bl + bd * dot(p - bl, bd), p);
}

__device__ float sd_ray(const vec3 p, const vec3 bl, const vec3 bd, float len) {
    float d = dot(p - bl, bd);

    if (d < 0) {
        return distance(bl, p);
    } else if (d > len) {
        return distance(bl + len * bd, p);
    }

    return distance(bl + bd * d, p);
}

__device__ float sd_line(const vec3 p, const vec3 b0, const vec3 b1) {
    float len = length(b1 - b0);
    return sd_ray(p, b0, (b1 - b0) / len, len);
}

__device__ float sd_unit_sphere(const vec3 p) {
    return length(p) - 0.5f;
}

__device__ float sd_box(const vec3 p, const vec3 bp, const vec3 bs) {
    vec3 q = abs(p - bp) - bs / 2.0f;
    float udst = length(max(q, vec3(0.0f)));
    float idst = maximum(min(q, vec3(0.0f)));
    return udst + idst;
}

__device__ float sd_box_skeleton(const vec3 p, const vec3 bp, const vec3 bs, const float lw) {
    vec3 bpl = bp - bs / 2.0f;
    float sd = MAX_POSITIVE_F32;

    for (int dir = 0; dir < 3; dir++) {
        for (int c0 = 0; c0 < 2; c0++) {
            for (int c1 = 0; c1 < 2; c1++) {
                vec3 m0 = bpl;
                m0[(dir + 1) % 3] += c0 ? bs[(dir + 1) % 2] : 0.0f;
                m0[(dir + 2) % 3] += c1 ? bs[(dir + 2) % 3] : 0.0f;

                vec3 m1 = m0;
                m1[dir] += bs[dir];

                sd = min(sd, sd_line(p, m0, m1) - lw);
            }
        }
    }

    return sd;
}

__device__ float sd_simple_box(const vec3 p, const vec3 bp, const vec3 bs) {
    vec3 q = abs(p - bp) - bs / 2.0f;
    return maximum(min(q, vec3(0.0f)));
}

__device__ float sd_simple_bounding_box(const vec3 p, const vec3 bb_min, const vec3 bb_max) {
    return max(
        max(
            bb_min.x - p.x,
            max(bb_min.y - p.y, bb_min.z - p.z)
        ),
        max(
            p.x - bb_max.x,
            max(p.y - bb_max.y, p.z - bb_max.z)
        )
    );
}

__device__ float sd_unit_cube(const vec3 p) {
    return sd_box(p, vec3(0.0f), vec3(1.0f));
}

bool inside_aabb(const vec3 p, const vec3 bb_min, const vec3 bb_max) {
    return bb_min.x <= p.x && p.x <= bb_max.x && bb_min.y <= p.y && p.y <= bb_max.y && bb_min.z <= p.z &&
           p.z <= bb_max.z;
}

float ray_distance_to_bb(const Ray &ray, const vec3 &bb_min, const vec3 &bb_max) {
    if (inside_aabb(ray.origin, bb_min, bb_max)) {
        return 0.0f;
    }

    float tmin = std::numeric_limits<float>::lowest();
    float tmax = std::numeric_limits<float>::max();

    for (int i = 0; i < 3; ++i) {
        if (abs(ray.direction[i]) < std::numeric_limits<float>::epsilon()) {
            // Ray is parallel to the slab. No hit if origin not within slab
            if (ray.origin[i] < bb_min[i] || ray.origin[i] > bb_max[i])
                return std::numeric_limits<float>::max();
        } else {
            // Compute intersection t value of ray with near and far plane of slab
            float ood = 1.0f / ray.direction[i];
            float t1 = (bb_min[i] - ray.origin[i]) * ood;
            float t2 = (bb_max[i] - ray.origin[i]) * ood;

            // Make t1 be intersection with near plane, t2 with far plane
            if (t1 > t2) std::swap(t1, t2);

            // Compute the intersection of slab intersection intervals
            tmin = max(tmin, t1);
            tmax = min(tmax, t2);

            // Exit with no collision as soon as slab intersection becomes empty
            if (tmin > tmax) return std::numeric_limits<float>::max();
        }
    }

    // Ray intersects all 3 slabs. Return distance to first hit
    return tmin > 0 ? tmin : tmax;
}

// normals

#define NORMAL_EPSILON 0.001f

template<typename SFunc>
__device__ vec3 empirical_normal(
    const SFunc sd_func,
    const vec3 p
) {
    float dx = (-sd_func(p + vec3(2.0f * NORMAL_EPSILON, 0.0f, 0.0f)) +
                8.0f * sd_func(p + vec3(NORMAL_EPSILON, 0.0f, 0.0f)) -
                8.0f * sd_func(p + vec3(-NORMAL_EPSILON, 0.0f, 0.0f)) +
                sd_func(p + vec3(-2.0f * NORMAL_EPSILON, 0.0f, 0.0f)));

    float dy = (-sd_func(p + vec3(0.0f, 2.0f * NORMAL_EPSILON, 0.0f)) +
                8.0f * sd_func(p + vec3(0.0f, NORMAL_EPSILON, 0.0f)) -
                8.0f * sd_func(p + vec3(0.0f, -NORMAL_EPSILON, 0.0f)) +
                sd_func(p + vec3(0.0f, -2.0f * NORMAL_EPSILON, 0.0f)));

    float dz = (-sd_func(p + vec3(0.0f, 0.0f, 2.0f * NORMAL_EPSILON)) +
                8.0f * sd_func(p + vec3(0.0f, 0.0f, NORMAL_EPSILON)) -
                8.0f * sd_func(p + vec3(0.0f, 0.0f, -NORMAL_EPSILON)) +
                sd_func(p + vec3(0.0f, 0.0f, -2.0f * NORMAL_EPSILON)));

    return normalize(vec3(dx, dy, dz));
}

struct NormalPlane {
    vec3 up;
    vec3 forward;
    vec3 right;
};

template<typename SFunc>
__device__ NormalPlane empirical_normal_plane(
    const SFunc sd_func,
    const vec3 p
) {
    NormalPlane plane;

    plane.up = empirical_normal(sd_func, p);
    plane.right = cross(
        plane.up,
        abs(dot(plane.up, vec3 { 0.0f, 0.0f, 1.0f })) < 0.5f ? vec3 { 0.0f, 0.0f, 1.0f } : vec3 { 0.0f, 1.0f, 0.0f }
    );
    plane.forward = cross(plane.up, plane.right);

    return plane;
}

template<typename SFunc>
__device__ vec3 closest_surface_point(const SFunc sd_func, const vec3 p) {
    vec3 g = p;

    bool collision = false;
    for (int i = 0; !collision && i < 10000; i++) {
        float sd = sd_func(g);
        vec3 n = empirical_normal(sd_func, g);
        g -= sd * n;
        collision = abs(sd) <= 0.001f;
    }

    return g;
}
