#include "hip/hip_runtime.h"
#include "./common.cu"

__device__ float sd_scene(vec3 p) {
    float sd = MAX_POSITIVE_F32;

    sd = min(sd, sd_obj(p));
    sd = min(sd, sd_box(p, vec3 { 0.0f, MESH_GENERATION_BB_MIN[1] - 0.5f, 0.0f }, vec3 { 10.0f, 1.0f, 10.0f }));
    sd = min(
        sd,
        sd_box_skeleton(
            p,
            (MESH_GENERATION_BB_MIN + MESH_GENERATION_BB_MAX) / 2.0f,
            MESH_GENERATION_BB_MAX - MESH_GENERATION_BB_MIN,
            0.05f
        )
    );

    return sd;
}

extern "C" __global__ void compute_render(
    const RenderTexture render_texture,
    const GlobalsBuffer globals,
    const CameraBuffer camera,
    const BlockPartition partition
) {
    // calculate ray

    uvec2 texture_coord = render_texture_coord({ render_texture.size[0], render_texture.size[1] });

    if (texture_coord.y >= render_texture.size[1] || texture_coord.x >= render_texture.size[0]) {
        return;
    }

    u32 texture_index = index_2d(texture_coord, render_texture);

    vec2 ndc_coord = texture_to_ndc(
        texture_coord,
        { render_texture.size[0], render_texture.size[1] }
    );
    vec2 cam_coord = ndc_to_camera(
        ndc_coord, { render_texture.size[0], render_texture.size[1] }
    );

    // ray marching

    Ray ray {
        { camera.position[0], camera.position[1], camera.position[2] },
        camera_to_ray(
            cam_coord,
            camera,
            from_array(globals.render_screen_size),
            vec2(globals.render_texture_size[0], globals.render_texture_size[1])
        )
    };

    float cone_radius_at_unit = get_pixel_cone_radius(
        texture_coord, camera, render_texture,
        globals
    );

    auto sd_scene_with_partition = [&](vec3 p) {
        float sd = sd_scene(p);

        if (globals.show_partition) {
            vec3 block_size = (MESH_GENERATION_BB_MAX - MESH_GENERATION_BB_MIN) / (float) partition.factor;

            for (int i = 0; i < partition.base_length; i++) {
                /*for (int c1 = 0; c1 <= 1; c1++) {
                    for (int c2 = 0; c2 <= 1; c2++) {
                        for (int c3 = 0; c3 <= 1; c3++) {
                            vec3 q = from_point(partition.bases[i]);
                            q.x += c1 ? block_size.x : 0.0f;
                            q.y += c2 ? block_size.y : 0.0f;
                            q.z += c3 ? block_size.z : 0.0f;
                            sd = min(sd, length(p - q) - 0.01f);
                        }
                    }
                }*/
                sd = min(sd, sd_box(p, from_point(partition.bases[i]) + block_size / 2.0f, block_size));
            }
        }

        return sd;
    };

    RayMarchHit hit = ray_march(sd_scene_with_partition, ray, cone_radius_at_unit);

    vec3 light_dir = normalize(vec3(1.0f, 1.0f, 1.0f));

    vec3 color { 0.0f };
    switch (hit.outcome) {
        case RayMarchHitOutcome::Collision: {
            vec3 normal = empirical_normal(sd_scene_with_partition, hit.position);
            color = mix(
                vec3 { 19.0f, 9.0f, 130.0f } / 255.0f,
                vec3 { 240.0f, 103.0f, 24.0f } / 255.0f,
                (dot(normal, light_dir) + 1.0f) / 2.0f
            );
            break;
        }

        case RayMarchHitOutcome::StepLimit:
            color = vec3 { 1.0f };
            break;

        case RayMarchHitOutcome::DepthLimit:
            break;
    }

    color = hdr_map_aces_tone(color);

    render_texture.data[texture_index] = {
        (unsigned char) (clamp(color.x, 0.0f, 1.0f) * 255.0f),
        (unsigned char) (clamp(color.y, 0.0f, 1.0f) * 255.0f),
        (unsigned char) (clamp(color.z, 0.0f, 1.0f) * 255.0f),
        0xFF,
    };
}
